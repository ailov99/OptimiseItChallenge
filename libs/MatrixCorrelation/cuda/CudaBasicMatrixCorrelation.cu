#include "hip/hip_runtime.h"
#ifdef __HIPCC__

#include <vector>
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>

// ======================= ERROR HANDLING =======================
static void HandleError(hipError_t err, const char *file, int line ) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),
                file, line );
        exit( EXIT_FAILURE );
    }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

// ====================== CUDA ===========================
__global__ 
void mykernel(float *normalised_m, float *result, int nx, int ny) {
    const auto x_dim = threadIdx.x + blockIdx.x * blockDim.x;
    const auto y_dim = threadIdx.y + blockIdx.y * blockDim.y;

    if (x_dim >= ny || y_dim >= ny)
        return;

    auto coeff = 0.f;
    for (auto x = 0; x < nx; x++)
        coeff += normalised_m[x + y_dim*nx] * normalised_m[x + x_dim*nx];
    result[x_dim*ny + y_dim] = coeff;
}


/*
Generate a correlation matrix of the input data.

@in
- ny = number of rows
- nx = number of columns
- data = input matrix
Note: Element at row y and column x is at data[x + y*nx]

@out
- result = output correlation matrix
Note: Correlation between rows i and row j is in result[i + j*ny]
Note: Only locations where 0 <= j <= i < ny are present (avoid redundancy)
*/
void correlate(int ny, int nx, const float *data, float *result) {
    float *normalised_m_host = new float[ny*nx];
    
    // Sum-norm to 0 and then to 1
    for (auto y = 0; y < ny; y++) {
        const auto *row_start = &(data[y*nx]);

        auto row_sum = 0.f;
        for (auto x = 0; x < nx; x++)
            row_sum += row_start[x];
        const auto row_mean = row_sum/nx;

        auto row_sq_sum = 0.f;
        for (auto x = 0; x < nx; x++)
            row_sq_sum += (row_start[x]-row_mean)*(row_start[x]-row_mean);
        const auto row_factor = std::sqrt(row_sq_sum);

        for (auto x = 0; x < nx; x++)
            normalised_m_host[x + y*nx] = (row_start[x] - row_mean) / row_factor;
    }

    float *normalised_m_dev = nullptr;
    HANDLE_ERROR(
        hipMalloc(
            (void**)&normalised_m_dev, 
            sizeof(float) * nx * ny
        )
    );
    float *result_buffer_dev = nullptr;
    HANDLE_ERROR(
        hipMalloc(
            (void**)&result_buffer_dev, 
            sizeof(float) * ny * ny
        )
    );
    HANDLE_ERROR(
        hipMemcpy(
            normalised_m_dev,
            normalised_m_host,
            sizeof(float) * nx * ny,
            hipMemcpyHostToDevice
        )
    );

    dim3 dimBlock(16, 16);
    dim3 dimGrid( (ny + dimBlock.x - 1)/dimBlock.x , (ny + dimBlock.y - 1)/dimBlock.y );
    mykernel<<<dimGrid, dimBlock>>>(normalised_m_dev, result_buffer_dev, nx, ny);
    HANDLE_ERROR(hipDeviceSynchronize());

    HANDLE_ERROR(
        hipMemcpy(
            result,
            result_buffer_dev,
            sizeof(float) * ny * ny,
            hipMemcpyDeviceToHost
        )
    );

    HANDLE_ERROR(hipFree(normalised_m_dev));
    HANDLE_ERROR(hipFree(result_buffer_dev));
    delete[] normalised_m_host;
}

#endif