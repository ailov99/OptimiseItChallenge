#include "hip/hip_runtime.h"
#ifdef __HIPCC__

#include <vector>
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>

// ======================= ERROR HANDLING =======================
static void HandleError(
    hipError_t err, 
    const char *file, 
    int line 
) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),
                file, line );
        exit( EXIT_FAILURE );
    }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

// ====================== CUDA ===========================
#define CHUNK_SIZE 64
#define CHUNK_DIM 8

__global__ 
void kernel_multiply(
    const float *input_m, 
    float *result, 
    const int nx, 
    const int ny,
    const int chunks_per_col
) {
    if (blockIdx.x > blockIdx.y)
        return;

    float chunk[CHUNK_SIZE];
    memset(chunk, 0.f, sizeof(chunk));
    
    for (auto x = 0; x < nx; x++) {
        float row[CHUNK_DIM];
        float col[CHUNK_DIM];

        for (auto i = 0; i < CHUNK_DIM; i++) {
            const auto x_offset = blockIdx.x*CHUNK_SIZE + i*CHUNK_DIM + threadIdx.x;
            const auto y_offset = blockIdx.y*CHUNK_SIZE + i*CHUNK_DIM + threadIdx.y;

            row[i] = input_m[chunks_per_col*x + x_offset];
            col[i] = input_m[chunks_per_col*x + y_offset];
        }

        for (auto i = 0; i < CHUNK_DIM; i++)
            for (auto j = 0; j < CHUNK_DIM; j++)
                chunk[CHUNK_DIM*i + j] += row[i] * col[j];
    }

    for (auto x = 0; x < CHUNK_DIM; x++) {
        for (auto y = 0; y < CHUNK_DIM; y++) {
            const auto real_x = CHUNK_SIZE*blockIdx.x + CHUNK_DIM*x + threadIdx.x;
            const auto real_y = CHUNK_SIZE*blockIdx.y + CHUNK_DIM*y + threadIdx.y;
            const auto in_result = (real_x < ny) && (real_y < ny);
            if (in_result)
                result[real_y + real_x*ny] = chunk[y + x*CHUNK_DIM];
        }
    }
}

__global__
void kernel_normalise(
    const float *input_m, 
    float *normalised_m, 
    const int nx, 
    const int ny
) {
    const auto y = CHUNK_SIZE * blockIdx.y + threadIdx.y;

    if (y >= ny)
        return;
    
    const auto *row_start = &(input_m[y*nx]);    

    auto row_sum = 0.f;
    for (auto x = 0; x < nx; x++)
        row_sum += row_start[x];
    const auto row_mean = row_sum/nx;

    auto row_sq_sum = 0.f;
    for (auto x = 0; x < nx; x++)
        row_sq_sum += (row_start[x]-row_mean)*(row_start[x]-row_mean);
    const auto row_factor = std::sqrt(row_sq_sum);

    for (auto x = 0; x < nx; x++)
        normalised_m[x + y*nx] = (row_start[x] - row_mean) / row_factor;
}

__global__
void kernel_xpose(
    const float *input_m,
    float *xposed_m,
    const int nx,
    const int ny,
    const int chunks_per_row,
    const int chunks_per_col
) {
    for (auto x = 0; x < chunks_per_row; x += CHUNK_SIZE) {
        const auto real_x = (x + threadIdx.x);
        const auto real_y = blockIdx.y;

        const auto xposed_index = real_x*chunks_per_col + real_y;

        const auto into_padding = (real_y >= ny) || (real_x >= nx);
        if (into_padding)
            xposed_m[xposed_index] = 0.f;
        else
            xposed_m[xposed_index] = input_m[nx*real_y + real_x];
    }
}


/*
Generate a correlation matrix of the input data.
This implementation takes advantage of all optimisations implemented so far: ILP, OMP, and Vectorisation.
Vectorised types are single-precision.

@in
- ny = number of rows
- nx = number of columns
- data = input matrix
Note: Element at row y and column x is at data[x + y*nx]

@out
- result = output correlation matrix
Note: Correlation between rows i and row j is in result[i + j*ny]
Note: Only locations where 0 <= j <= i < ny are present (avoid redundancy)
*/
void correlate(
    int ny, 
    int nx, 
    const float *data, 
    float *result
) {
    // Init
    const auto chunks_per_row = ((nx + CHUNK_SIZE - 1) / CHUNK_SIZE) * CHUNK_SIZE;
    const auto chunks_per_col = ((ny + CHUNK_SIZE - 1) / CHUNK_SIZE) * CHUNK_SIZE;

    float *m_dev = nullptr;
    float *m_dev_norm = nullptr;

    HANDLE_ERROR(
        hipMalloc(
            (void**)&m_dev_norm, 
            sizeof(float) * nx * ny
        )
    );
    HANDLE_ERROR(
        hipMalloc(
            (void**)&m_dev, 
            sizeof(float) * nx * ny
        )
    );
    HANDLE_ERROR(
        hipMemcpy(
            m_dev,
            data,
            sizeof(float) * nx * ny,
            hipMemcpyHostToDevice
        )
    );

    // Norm (element-by-element)
    dim3 norm_dimBlock(1, CHUNK_SIZE);
    dim3 norm_dimGrid(1, chunks_per_col/CHUNK_SIZE);
    kernel_normalise<<<norm_dimGrid, norm_dimBlock>>>(m_dev, m_dev_norm, nx, ny);
    HANDLE_ERROR(hipDeviceSynchronize());

    // Transpose (in chunks)
    float *m_dev_xposed = nullptr;
    HANDLE_ERROR(
        hipMalloc(
            (void**)&m_dev_xposed,
            sizeof(float) * chunks_per_row * chunks_per_col
        )
    );
    dim3 xpose_dimBlock(CHUNK_SIZE, 1);
    dim3 xpose_dimGrid(1, chunks_per_col);
    kernel_xpose<<<xpose_dimGrid, xpose_dimBlock>>>(m_dev_norm, m_dev_xposed, nx, ny, chunks_per_row, chunks_per_col);
    HANDLE_ERROR(hipDeviceSynchronize());


    // Multiply (in chunks)
    float *result_buffer_dev = nullptr;
    HANDLE_ERROR(
        hipMalloc(
            (void**)&result_buffer_dev, 
            sizeof(float) * ny * ny
        )
    );
    HANDLE_ERROR(
        hipMemset(
            result_buffer_dev,
            0.f,
            sizeof(float) * ny * ny
        )
    );

    dim3 mult_dimBlock(CHUNK_DIM, CHUNK_DIM);
    dim3 mult_dimGrid((chunks_per_col / CHUNK_SIZE),(chunks_per_col / CHUNK_SIZE));
    kernel_multiply<<<mult_dimGrid, mult_dimBlock>>>(m_dev_xposed, result_buffer_dev, nx, ny, chunks_per_col);
    HANDLE_ERROR(hipDeviceSynchronize());

    HANDLE_ERROR(
        hipMemcpy(
            result,
            result_buffer_dev,
            sizeof(float) * ny * ny,
            hipMemcpyDeviceToHost
        )
    );

    HANDLE_ERROR(hipFree(m_dev));
    HANDLE_ERROR(hipFree(m_dev_norm));
    HANDLE_ERROR(hipFree(m_dev_xposed));
    HANDLE_ERROR(hipFree(result_buffer_dev));
}

#endif